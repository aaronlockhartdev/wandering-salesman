#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

#define NUM_QUEUES 1024
#define SIZE 10

const int START[2] = {0, 0};

typedef struct Node Node;

struct Node
{
    float f;
    float g;
    int x;
    int y;
    Node * parent;
    Node * next;
    Node * prev;
};

__constant__ int directions_c[8][2];
__constant__ int goal_c[2];

__global__ void initBoard(float[][] board)
{
    #include<math.h>
    #include<time.h>

    srand(clock() + blockIdx.x + blockIdx.y * blockDim.x);
    board[blockIdx.x][blockIdx.y] = ((float)(rand() % 100 + 1)) / 100;
    printf("%f\n", board[blockIdx.x][blockIdx.y]);
}

void printBoard(float[][] board)
{
    for (int x = 0; x < SIZE; ++x)
    {
        for (int y = 0; y < SIZE; ++y)
        {
            printf("%d ", (int)(board[x][y] * 100));
        }
        printf("\n");
    }
}

int main()
{
    int directions_h[8][2] = {{1, 1}, {1, 0}, {1, -1}, {0, 1}, {0, -1}, {-1, 1}, {-1, 0}, {-1, -1}};
    int goal_h[2] = {SIZE - 1, SIZE - 1};

    hipMemcpyToSymbol(HIP_SYMBOL(directions_c), directions_h, sizeof(int) * 16);
    hipMemcpyToSymbol(HIP_SYMBOL(goal_c), goal_h, sizeof(int) * 2);

    float board[SIZE][SIZE];
    hipMallocManaged(&board, sizeof(float) * SIZE * SIZE);

    initBoard<<<SIZE, SIZE, 1>>>(board);
    hipDeviceSynchronize();
    printBoard(board);

    Node * start;
    Node * closed;
    Node * best;
    Node* open[NUM_QUEUES];
    float hashTable[SIZE][SIZE];

    hipMallocManaged(&start, sizeof(Node));
    hipMallocManaged(&closed, sizeof(Node));
    hipMallocManaged(&best, sizeof(Node));
    hipMallocManaged(&open, sizeof(Node*) * NUM_QUEUES);
    hipMallocManaged(&hashTable, sizeof(float) * SIZE * SIZE);

    start->f = 0;
    start->g = 0;
    start->x = START[0];
    start->y = START[1];
    start->parent = NULL;
    start->next = NULL;
    start->prev = NULL;


    hipFree(start);

    return 0;
}